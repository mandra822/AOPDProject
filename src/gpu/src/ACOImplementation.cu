#include "hip/hip_runtime.h"
﻿#include "AODPProject/gpu/ACOImplementation.h"
#include <thread>
#include <mutex>
#include <vector>
#include <algorithm>
#include <cstring>

#include <hip/hip_runtime.h>
#include ""
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ float calculateDenominator(
    float* chances, int visitedCount,
    int numberOfVertexes) {

    float denominator = 0;

    for (int i = visitedCount; i < numberOfVertexes; i++) {
        denominator += chances[i];
    }

    return denominator;
}

__global__ void calculateProbability(
    float* resultProbability, 
    float* pheromoneMatrix, int* edgesMatrix, int numberOfVertexes) {

    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadId > numberOfVertexes * numberOfVertexes) return;
    float alpha = 1.0f, beta = 3.0f; // Example parameters

    float edgeCost = edgesMatrix[threadId];
    float nominator = 0;
    if (edgeCost != 0) {
        nominator = powf(pheromoneMatrix[threadId], alpha) * powf(1.0f / edgeCost, beta);
    }
    else {
        nominator = powf(pheromoneMatrix[threadId], alpha) * powf(1.0f / 0.1f, beta);
    }
    resultProbability[threadId] = nominator;
}

__global__ void evapouratePheromoneD(float* pheromoneMatrix, float rate,int  numberOfVertexes){

    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadId > numberOfVertexes * numberOfVertexes) return;
    pheromoneMatrix[threadId] *= rate;
}
__global__ void leavePheromone(float* pheromoneMatrix, int* edgesMatrix, int* ants, int numberOfVertexes, float Qcycl, int* costs, int colonySize) {
    
    int threadId = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadId > colonySize) return;
    auto cost = 0;
    for(auto i = 1; i < numberOfVertexes; i++) {
        cost += edgesMatrix[ants[threadId * numberOfVertexes +i-1] * numberOfVertexes + ants[threadId * numberOfVertexes +i]];
    }
    costs[threadId] = cost;
    for(auto i = 1; i < numberOfVertexes; i++) {
        pheromoneMatrix[ants[threadId * numberOfVertexes +i-1] * numberOfVertexes + ants[i]] += (float)Qcycl/(float)cost;
    }
}

__device__ int choseVertexByProbability(
    int* sharedInt, float* chances, int visitedCount, int numberOfVertexes, hiprandState &state) {

    float toss = hiprand_uniform_double(&state), cumulativeSum = 0.0f;

    for (int i = visitedCount; i < numberOfVertexes; i++) {

        cumulativeSum += chances[i];
        if (!(cumulativeSum == cumulativeSum) || cumulativeSum > toss) {
            return sharedInt[i];
        }
    }

    return sharedInt[numberOfVertexes - 1]; // Fallback in case of numerical issues
}

__device__ void calculateNominatorToShared(
    int* vertexes,
    float* chances, 
    int ownVertex,
    int position,
    int prevVertex,
    float* pheromoneMatrix, int* edgesMatrix, int numberOfVertexes) {

    float alpha = 1.0f, beta = 3.0f; // Example parameters

    int edgeCost = edgesMatrix[ownVertex];
    float nominator = 0.0f;
    if (edgeCost != 0) {
        nominator = (float)std::pow(pheromoneMatrix[prevVertex * numberOfVertexes + ownVertex], alpha) * std::pow(1.0f / edgeCost, beta);
    }
    else {
        nominator = (float)std::pow(pheromoneMatrix[prevVertex * numberOfVertexes + ownVertex], alpha) * std::pow(1.0f / 0.1f, beta);
    }
    chances[position] = nominator;
    vertexes[position] = ownVertex;
}
__device__ void normalize(
    float* chances,
    float* denominator,
    int position
        ){
    chances[position] /= *denominator;
}
__global__ void findSolutions(int* solutionsPointer, int* edgesMatrix, float* pheromoneMatrix, int numberOfVertexes) {

    extern __shared__ int sharedInt[];
    size_t shOffset = (sizeof(float)/sizeof(int)*(numberOfVertexes));
    float* sharedFloat = (float*)(&sharedInt[shOffset]);
    float* denominator = &sharedFloat[numberOfVertexes];

    int threadId = blockIdx.x;

    //int* vertices = &sharedInt[numberOfVertexes * threadId]; 
    //float* chances = &sharedFloat[threadId * numberOfVertexes];

    //float* chances;
    //int* vertices;
    //hipMalloc(&chances, numberOfVertexes * sizeof(float));
    //hipMalloc(&vertices, numberOfVertexes * sizeof(int));

    hiprandState state;
    hiprand_init((unsigned long long)clock() + threadId, 0, 0, &state);

    // Each thread handles one solution
    int* solution = &solutionsPointer[threadId * numberOfVertexes];
    int lastVisitedVertex = (int)(hiprand_uniform(&state) * numberOfVertexes);
    solution[0] = lastVisitedVertex;
    sharedInt[0] = lastVisitedVertex;

    int visitedCount = 1;

    auto skip = false;
 
    if (threadIdx.x > numberOfVertexes + 1 || threadId > numberOfVertexes) skip = true;
    auto position = threadIdx.x;
    auto ownVertex = threadIdx.x;
    float alpha = 1.0f, beta = 3.0f; // Example parameters
    int nextVertex;
    while (visitedCount < numberOfVertexes) {
        if (threadIdx.x != 0 && !skip) {
            //printf("N1 thread %d block %d\n", threadIdx.x, blockIdx.x);
            auto prev = sharedInt[visitedCount-1];
            if (ownVertex == prev) skip = true;
            else if (prev > ownVertex) position++;
            if (!skip) calculateNominatorToShared(sharedInt, sharedFloat, ownVertex, position, prev, pheromoneMatrix, edgesMatrix, numberOfVertexes);
        }
        __syncthreads();
        if (threadIdx.x == 0 && !skip) {
            //printf("A1 thread %d block %d\n", threadIdx.x), blockIdx.x;
            *denominator = calculateDenominator( sharedFloat, visitedCount, numberOfVertexes);
        }
        __syncthreads();
        if (threadIdx.x != 0 && !skip) {
            //printf("N2 thread %d block %d\n", threadIdx.x), blockIdx.x;
            normalize(sharedFloat, denominator, position);
        }
        __syncthreads();
        if (threadIdx.x == 0 && !skip) {
            //printf("A2 thread %d block %d\n", threadIdx.x), blockIdx.x;
            nextVertex = choseVertexByProbability(sharedInt, sharedFloat, visitedCount, numberOfVertexes, state);
            sharedInt[visitedCount] = nextVertex;
        }
        __syncthreads();
        lastVisitedVertex = nextVertex;
        visitedCount++;

        if (threadIdx.x == 0 && !skip) {
            //printf("A3 thread %d block %d\n", threadIdx.x, blockIdx.x);
            solution[visitedCount] = nextVertex;
        }
    }

    //hipFree(chances);
    //hipFree(vertices);
}

namespace GPU {

    void ACOImplementation::init(int startingVertex, std::vector<std::vector<int>> edges, float alpha, float beta, int numberOfVertexes, int colonySize)
    {
        this->startingVertex = startingVertex;
        this->edges = edges;
        this->alpha = alpha;
        this->beta = beta;
        this->colonySize = colonySize;
        this->numberOfVertexes = numberOfVertexes;
        this->result = (int*)malloc(numberOfVertexes * sizeof(int));
        initializePheromoneMatrix(calculateApproximatedSolutionCost());
    }

    int* ACOImplementation::runAcoAlgorith(int numberOfIterations)
    {
        int startingVertexForAnt = startingVertex;
        int chosenVertex;


        //Copy eges and pheromone matrix into GPU memory
        std::vector<int> flatEdges;
        for (const auto& row : edges) {
            flatEdges.insert(flatEdges.end(), row.begin(), row.end());
        }

        int* d_edges;
        hipMalloc(&d_edges, flatEdges.size() * sizeof(int));
        hipMemcpy(d_edges, flatEdges.data(), flatEdges.size() * sizeof(int), hipMemcpyHostToDevice);

        std::vector<float> flatPheromone;
        for (const auto& row : pheromoneMatrix) {
            flatPheromone.insert(flatPheromone.end(), row.begin(), row.end());
        }

        float* d_pheromoneMatrix;
        hipMalloc(&d_pheromoneMatrix, flatPheromone.size() * sizeof(float));
        hipMemcpy(d_pheromoneMatrix, flatPheromone.data(), flatPheromone.size() * sizeof(float), hipMemcpyHostToDevice);

        float* d_probMatrix;
        hipMalloc(&d_probMatrix, numberOfVertexes * numberOfVertexes * sizeof(float));

        int* d_colony;
        hipMalloc(&d_colony, colonySize * edges.size() * sizeof(int));

        int* h_costs = (int*)malloc(colonySize * sizeof(int));
        for (auto i = 0; i < numberOfVertexes; i++) {
            h_costs[i] = i;
        }
        int* d_costs;
        hipMalloc(&d_costs, colonySize * sizeof(int));

        int* h_colony = (int*)malloc(colonySize * edges.size() * sizeof(int));
        for (int j = 0; j < numberOfIterations; j++) {
            
            /*for (int i = 0; i < colonySize; i++) {
                while (startingVertexForAnt == startingVertex) {
                    startingVertexForAnt = rand() % edges.size();
                }
                h_colony[i*edges.size()] = startingVertex;
                startingVertexForAnt = startingVertex;
            }*/
            
            //hipMemcpy(d_colony, h_colony, colonySize * sizeof(int**), hipMemcpyHostToDevice);
            //hipMalloc(&d_solutions, colonySize * sizeof(int));
            //hipMemcpy(d_solutions, colony.data(), colony.size() * sizeof(int), hipMemcpyHostToDevice);
            int blockMaxSize = -1;
            int threadsPerBlock = 32;
            //int numberOfBlocks = colonySize/threadsPerBlock + 1;

            //int sharedMemorySize = threadsPerBlock * numberOfVertexes * (sizeof(float) + sizeof(int));
            int sharedMemorySize = blockMaxSize;
            //int threadsPerBlock =  sharedMemorySize / (numberOfVertexes * (sizeof(float) + sizeof(int)));
            int numberOfBlocks = colonySize/threadsPerBlock + 1;


            //int* antsData;
            //hipMalloc(&antsData, numberOfVertexes * colonySize * (sizeof(float) + sizeof(int)));

            //sharedMemorySize += sizeof(int) - sharedMemorySize % sizeof(int);
            //printf("Colony size is: %d\nNumber of vertices: %d\nBlock max shared mem size: %d\nStarting Kernel on %d blocks each %d threads with %d bytes of shared memory\n", colonySize, numberOfVertexes, blockMaxSize, numberOfBlocks, threadsPerBlock, sharedMemorySize);
            //do dopracowania (1 oznacza ilosc blokow, 1024 ilosc watkow na blok)
            findSolutions <<<colonySize, numberOfVertexes + 1, numberOfVertexes * (sizeof(float) + sizeof(int)) + sizeof(float) >>> (d_colony, d_edges, d_pheromoneMatrix, numberOfVertexes );
            evapouratePheromoneD<<<numberOfVertexes * numberOfVertexes / threadsPerBlock, threadsPerBlock>>>(d_pheromoneMatrix, 0.1, numberOfVertexes);
            leavePheromone <<<numberOfBlocks, threadsPerBlock>>> (d_pheromoneMatrix,  d_edges, d_colony, numberOfVertexes, 0.4, d_costs, colonySize); 

            //hipFree(antsData);
            //hipMemcpy(colony.data(), d_solutions, colony.size() * sizeof(int), hipMemcpyDeviceToHost);
            //TUTAJ są kopiuowane tylko wskaźniki do tablic z rozwiazaniami mrówek a nie same ścieżki z mrówkami

            hipMemcpy(h_costs, d_costs, colonySize * sizeof(int), hipMemcpyDeviceToHost);
            int bestIndex = -1;
            for(auto i = 0; i < colonySize; i++) {
                if(h_costs[i] < minCost) {
                    minCost = h_costs[i];
                    bestIndex = i;
                }
            }
            if (bestIndex != -1) {
                hipMemcpy(result, &d_colony[bestIndex * numberOfVertexes], numberOfVertexes * sizeof(int), hipMemcpyDeviceToHost);
            }
            


            //evaporation
        }
        return result;
    }

    bool containsOnlyCities(int* path, int numberOfCities) {
        auto zero_corrected = false;
        for(auto i = 0; i < numberOfCities; i++) {
            if(path[i] > numberOfCities || path[i] < 0) {
                if (zero_corrected) return false;
                zero_corrected = true;
                path[i] = 0;
            }
        }
        return true;
    }

    void ACOImplementation::evaporatePheromoneCAS(float Qcycl, float pheromoneEvaporationRate, int* colony)
    {
        int cost;

        evaporatePheromone(pheromoneEvaporationRate);
        int* antSolution;
        int* _result = nullptr;
        for (auto ant = 0; ant < colonySize; ant++)
        {
            antSolution = &colony[ant * edges.size()];
            //if(containsOnlyCities(antSolution, numberOfVertexes)) continue;
            cost = calculateSolutionCost(antSolution);
            if (cost < minCost)
            {
                minCost = cost;
                _result = antSolution;
            }

            for (int i = 0; i < numberOfVertexes-1; i++)
            {
                pheromoneMatrix[antSolution[i]][antSolution[i + 1]] += (float)Qcycl / cost;
            }
        }
        if (_result != nullptr) {
            std::memcpy(result, _result, numberOfVertexes * sizeof(int));
        }
    }

    void ACOImplementation::evaporatePheromone(float pheromoneEvaporationRate)
    {
        for (int i = 0; i < numberOfVertexes; i++)
        {
            for (int j = 0; j < numberOfVertexes; j++)
            {
                pheromoneMatrix[i][j] *= pheromoneEvaporationRate;
            }
        }
    }

    int ACOImplementation::calculateSolutionCost(int* solution)
    {
        int cost = 0;
        for (int i = 0; i < numberOfVertexes-1; i++)
        {
            cost += edges[solution[i]][solution[i + 1]];
        }

        cost += edges[startingVertex][solution[0]];					
        cost += edges[solution[numberOfVertexes-1]][startingVertex];

        return cost;
    }

    void ACOImplementation::initializePheromoneMatrix(int aproximatedSolutionCost)
    {
        float tau_zero = (float)colonySize / (float)aproximatedSolutionCost;
        std::vector<float> tempVec;

        for (int i = 0; i < numberOfVertexes; i++)
        {
            tempVec.push_back(tau_zero);
        }

        for (int i = 0; i < numberOfVertexes; i++)
        {
            pheromoneMatrix.push_back(tempVec);
        }
    }

    float ACOImplementation::calculateApproximatedSolutionCost()
    {
        int* solution = new int[numberOfVertexes];

        int randIndexI, randIndexJ;

        for (int i = 0; i < numberOfVertexes; i++) solution[i] = i;

        for (int i = 0; i < numberOfVertexes; i++)
        {
            randIndexI = rand() % numberOfVertexes;	// toss index (0 , solution-1)
            randIndexJ = rand() % numberOfVertexes;
            std::swap(solution[randIndexI], solution[randIndexJ]);
        }

        //Divide value as there is high probability that this is not even close 
        //to the optimal value
        return calculateSolutionCost(solution) * 0.6f;
    }
}

